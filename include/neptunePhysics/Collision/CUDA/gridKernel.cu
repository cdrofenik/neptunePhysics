#include "hip/hip_runtime.h"
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

namespace NeptunePhysics
{
	__global__ void AddIntsCUDA(int *a, int *b)
	{
		a[0] += b[0];
	}

	extern "C" int cuda_func(int &a, const int &b)
	{
		int *dev_a, *dev_b;

		hipMalloc(&dev_a, sizeof(int));
		hipMalloc(&dev_b, sizeof(int));

		hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);

		AddIntsCUDA << <1, 1 >> >(dev_a, dev_b);

		hipMemcpy(&a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

		std::cout << "The result is: " << a << std::endl;

		// free the memory allocated on the GPU
		hipFree(&dev_a);
		hipFree(&dev_b);

		return a;
	}
}