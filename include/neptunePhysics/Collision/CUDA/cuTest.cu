#include "hip/hip_runtime.h"
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

using namespace std;

__global__ void AddIntsCUDA(int *a, int *b)
{
	a[0] += b[0];
}

int main()
{
	int a = 5, b = 9;
	int *dev_a, *dev_b;

	hipMalloc(&dev_a, sizeof(int));
	hipMalloc(&dev_b, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCUDA << <1, 1 >> >(dev_a, dev_b);

	hipMemcpy(&a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

	cout << "The result is: " << a << endl;

	// free the memory allocated on the GPU
	hipFree(&dev_a);
	hipFree(&dev_b);

	return 0;
}